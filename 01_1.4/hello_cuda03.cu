
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}