
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU!\n");
}

int main(void)
{
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}