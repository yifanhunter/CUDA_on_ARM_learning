#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdint.h>
#include<time.h>     //for time()
#include<stdlib.h>   //for srand()/rand()
#include<sys/time.h> //for gettimeofday()/struct timeval
#include"error.cuh"

#define N 10000000
#define BLOCK_SIZE 256
#define BLOCKS ((N + BLOCK_SIZE - 1) / BLOCK_SIZE) 


__managed__ int source[N];               //input data
__managed__ int final_result[1] = {0};   //scalar output

__global__ void _sum_gpu(int *input, int count, int *output)
{
    __shared__ int sum_per_block[BLOCK_SIZE];

    int temp = 0;
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
         idx < count;
	 idx += gridDim.x * blockDim.x
	)
    {
        temp += input[idx];
    }

    sum_per_block[threadIdx.x] = temp;  //the per-thread partial sum is temp!
    __syncthreads();

    //**********shared memory summation stage***********
    for (int length = BLOCK_SIZE / 2; length >= 1; length /= 2)
    {
        int double_kill = -1;
	if (threadIdx.x < length)
	{
	    double_kill = sum_per_block[threadIdx.x] + sum_per_block[threadIdx.x + length];
	}
	__syncthreads();  //why we need two __syncthreads() here, and,
	
	if (threadIdx.x < length)
	{
	    sum_per_block[threadIdx.x] = double_kill;
	}
	__syncthreads();  //....here ?
	
    } //the per-block partial sum is sum_per_block[0]

    if (blockDim.x * blockIdx.x < count) //in case that our users are naughty
    {
        //the final reduction performed by atomicAdd()
        if (threadIdx.x == 0) atomicAdd(output, sum_per_block[0]);
    }
}

int _sum_cpu(int *ptr, int count)
{
    int sum = 0;
    for (int i = 0; i < count; i++)
    {
        sum += ptr[i];
    }
    return sum;
}

void _init(int *ptr, int count)
{
    uint32_t seed = (uint32_t)time(NULL); //make huan happy
    srand(seed);  //reseeding the random generator

    //filling the buffer with random data
    for (int i = 0; i < count; i++) ptr[i] = rand();
}

double get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return ((double)tv.tv_usec * 0.000001 + tv.tv_sec);
}

int main()
{
    //**********************************
    fprintf(stderr, "filling the buffer with %d elements...\n", N);
    _init(source, N);

    //**********************************
    //Now we are going to kick start your kernel.
    hipDeviceSynchronize(); //steady! ready! go!
    
    fprintf(stderr, "Running on GPU...\n");
    
double t0 = get_time();
    _sum_gpu<<<BLOCKS, BLOCK_SIZE>>>(source, N, final_result);
    CHECK(hipGetLastError());  //checking for launch failures
    CHECK(hipDeviceSynchronize()); //checking for run-time failurs
double t1 = get_time();

    int A = final_result[0];
    fprintf(stderr, "GPU sum: %u\n", A);


    //**********************************
    //Now we are going to exercise your CPU...
    fprintf(stderr, "Running on CPU...\n");

double t2 = get_time();
    int B = _sum_cpu(source, N);
double t3 = get_time();
    fprintf(stderr, "CPU sum: %u\n", B);

    //******The last judgement**********
    if (A == B)
    {
        fprintf(stderr, "Test Passed!\n");
    }
    else
    {
        fprintf(stderr, "Test failed!\n");
	exit(-1);
    }
    
    //****and some timing details*******
    fprintf(stderr, "GPU time %.3f ms\n", (t1 - t0) * 1000.0);
    fprintf(stderr, "CPU time %.3f ms\n", (t3 - t2) * 1000.0);

    return 0;
}	
	
