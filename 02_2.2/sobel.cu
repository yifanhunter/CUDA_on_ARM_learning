#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>

using namespace std;
using namespace cv;

//GPU实现Sobel边缘检测
//             x0 x1 x2 
//             x3 x4 x5 
//             x6 x7 x8 
__global__ void sobel_gpu(unsigned char* in, unsigned char* out, int imgHeight, int imgWidth)
{

}

//CPU实现Sobel边缘检测
void sobel_cpu(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar* dataUp = srcImg.ptr<uchar>(i - 1);
        uchar* data = srcImg.ptr<uchar>(i);
        uchar* dataDown = srcImg.ptr<uchar>(i + 1);
        uchar* out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1])-(dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}

int main()
{
    //利用opencv的接口读取图片
    Mat img = imread("1.jpg", 0);
    int imgWidth = img.cols;
    int imgHeight = img.rows;

    //利用opencv的接口对读入的grayImg进行去噪
    Mat gaussImg;
    GaussianBlur(img, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);
    //CPU结果为dst_cpu, GPU结果为dst_gpu
    Mat dst_cpu(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    Mat dst_gpu(imgHeight, imgWidth, CV_8UC1, Scalar(0));


    //调用sobel_cpu处理图像
    sobel_cpu(gaussImg, dst_cpu, imgHeight, imgWidth);

    //申请指针并将它指向GPU空间
    size_t num = imgHeight * imgWidth * sizeof(unsigned char);
    unsigned char* in_gpu;
    unsigned char* out_gpu;
    hipMalloc((void**)&in_gpu, num);
    hipMalloc((void**)&out_gpu, num);
    //定义grid和block的维度（形状）
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //将数据从CPU传输到GPU
    hipMemcpy(in_gpu, img.data, num, hipMemcpyHostToDevice);
    //调用在GPU上运行的核函数
    sobel_gpu<<<blocksPerGrid,threadsPerBlock>>>(in_gpu, out_gpu, imgHeight, imgWidth);

    //将计算结果传回CPU内存
    hipMemcpy(dst_gpu.data, out_gpu, num, hipMemcpyDeviceToHost);
    imwrite("save.png", dst_gpu);
    //显示处理结果, 由于这里的Jupyter模式不支持显示图像, 所以我们就不显示了
    //imshow("gpu", dst_gpu);
    //imshow("cpu", dst_cpu);
    //waitKey(0);
    //释放GPU内存空间
    hipFree(in_gpu);
    hipFree(out_gpu);
    return 0;
}
