
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

void __global__ add(const double *x, const double *y, double *z, int count)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
	if( n < count)
	{
	    z[n] = x[n] + y[n];
	}

}
void check(const double *z, const int N)
{
    bool error = false;
    for (int n = 0; n < N; ++n)
    {
        if (fabs(z[n] - 3) > (1.0e-10))
        {
            error = true;
        }
    }
    printf("%s\n", error ? "Errors" : "Pass");
}


int main(void)
{
    const int N = 1000;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1;
        h_y[n] = 2;
    }

    double *d_x, *d_y, *d_z;
    hipMalloc((void **)&d_x, M);
    hipMalloc((void **)&d_y, M);
    hipMalloc((void **)&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}