#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void gpu_matrix_mult_shared(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = row<n && (sub * BLOCK_SIZE + threadIdx.x)<n? d_a[idx]:0;
        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        tile_b[threadIdx.y][threadIdx.x] = col<n && (sub * BLOCK_SIZE + threadIdx.y)<n? d_b[idx]:0;

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int m=1000;
    int n=1000;
    int k=1000;

    int *h_a, *h_b, *h_c, *h_cc, *h_cs;
    CHECK(hipHostMalloc((void **) &h_a, sizeof(int)*m*n));
    CHECK(hipHostMalloc((void **) &h_b, sizeof(int)*n*k));
    CHECK(hipHostMalloc((void **) &h_c, sizeof(int)*m*k));
    CHECK(hipHostMalloc((void **) &h_cc, sizeof(int)*m*k));
    CHECK(hipHostMalloc((void **) &h_cs, sizeof(int)*m*k));
    
    hipEvent_t start, stop,stop_share;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventCreate(&stop_share));


    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = 1;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = 0;
        }
    }

    int *d_a, *d_b, *d_c, *d_c_share;
    CHECK(hipMalloc((void **) &d_a, sizeof(int)*m*n));
    CHECK(hipMalloc((void **) &d_b, sizeof(int)*n*k));
    CHECK(hipMalloc((void **) &d_c, sizeof(int)*m*k));
    CHECK(hipMalloc((void **) &d_c_share, sizeof(int)*m*k));

    CHECK(hipEventRecord(start));
    // copy matrix A and B from host to device memory
    CHECK(hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice));

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m,n,k);    

    CHECK(hipMemcpy(h_c, d_c, (sizeof(int)*m*k), hipMemcpyDeviceToHost));
    //hipDeviceSynchronize();
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    
    gpu_matrix_mult_shared<<<dimGrid, dimBlock>>>(d_a, d_b, d_c_share, n);
    CHECK(hipMemcpy(h_cs, d_c_share, (sizeof(int)*m*k), hipMemcpyDeviceToHost));
    
    CHECK(hipEventRecord(stop_share));
    CHECK(hipEventSynchronize(stop_share));
    
    float elapsed_time, elapsed_time_share;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    CHECK(hipEventElapsedTime(&elapsed_time_share, stop, stop_share));
    printf("Time_global = %g ms.\n", elapsed_time);
    printf("Time_share = %g ms.\n", elapsed_time_share);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));    

    //cpu_matrix_mult(h_a, h_b, h_c, m, n, k);

    int ok = 1;
    for (int i = 0; i < m; ++i)
    { 
        for (int j = 0; j < k; ++j)
        {
            if(fabs(h_cs[i*k + j] - 0)>(1.0e-10))
            {
                printf("hcs: %d hc: %d  ",h_cs[i*k + j], h_c[i*k + j]);
                ok = 0;
            }
        }
    }

    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    
    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    CHECK(hipHostFree(h_a));
    CHECK(hipHostFree(h_b));
    CHECK(hipHostFree(h_c));
    return 0;
}