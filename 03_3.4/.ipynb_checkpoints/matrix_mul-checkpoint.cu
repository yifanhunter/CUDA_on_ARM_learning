#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE  8

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void gpu_matrix_mult_shared(int *d_a, int *d_b, int *d_result, int m, int n, int k) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE]; // 共享变量
 
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x; // 横纵坐标
    int tmp = 0;
    int idx;
 
    for (int sub = 0; sub <= n/BLOCK_SIZE; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x; // 共享变量的对应于原数据的一维下标
       // printf("%d %d \n",sub * BLOCK_SIZE + threadIdx.x,n);
        tile_a[threadIdx.y][threadIdx.x] = row<m && (sub * BLOCK_SIZE + threadIdx.x)<n? d_a[idx]:0; // 规避多余操作
        idx = (sub * BLOCK_SIZE + threadIdx.y) * k + col; //原来的代码是乘以n，需要改成K
       // printf("%d %d %d %d %d\n",col,k,sub * BLOCK_SIZE + threadIdx.y, n,idx);
        tile_b[threadIdx.y][threadIdx.x] = col<k && (sub * BLOCK_SIZE + threadIdx.y)<n? d_b[idx]:0;
        /*
        一个共享块的例子：
        针对于tile_a
        共享变量的下标                              原数据下标:
        (0,0) (0,1) ... (0,15)                    0 1 2 ... 15
        (1,0) ...       (1,15)     --------->     n n+1 ... n+15
        ...                                       ...
        (15,0) ...      (15,15)                      15n 15n+1 ... 15n+15
        */
        
        __syncthreads(); //同步上述操作 求和
        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < m && col < k)
    {
        d_result[row * k + col] = tmp;
    }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int m=1000;
    int n=1000;
    int k=1000;

    int *h_a, *h_b, *h_c, *h_cc, *h_cs;
    CHECK(hipHostMalloc((void **) &h_a, sizeof(int)*m*n));
    CHECK(hipHostMalloc((void **) &h_b, sizeof(int)*n*k));
    CHECK(hipHostMalloc((void **) &h_c, sizeof(int)*m*k));
    CHECK(hipHostMalloc((void **) &h_cc, sizeof(int)*m*k));
    CHECK(hipHostMalloc((void **) &h_cs, sizeof(int)*m*k));
    
    hipEvent_t start, stop,stop_share;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventCreate(&stop_share));
    
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }
    

    int *d_a, *d_b, *d_c, *d_c_share;
    CHECK(hipMalloc((void **) &d_a, sizeof(int)*m*n));
    CHECK(hipMalloc((void **) &d_b, sizeof(int)*n*k));
    CHECK(hipMalloc((void **) &d_c, sizeof(int)*m*k));
    CHECK(hipMalloc((void **) &d_c_share, sizeof(int)*m*k));

    CHECK(hipEventRecord(start));
    // copy matrix A and B from host to device memory
    CHECK(hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice));

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m,n,k);    

    CHECK(hipMemcpy(h_c, d_c, (sizeof(int)*m*k), hipMemcpyDeviceToHost));
    //hipDeviceSynchronize();
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    
    gpu_matrix_mult_shared<<<dimGrid, dimBlock>>>(d_a, d_b, d_c_share, m,n,k) ;
    CHECK(hipMemcpy(h_cs, d_c_share, (sizeof(int)*m*k), hipMemcpyDeviceToHost));
    
    CHECK(hipEventRecord(stop_share));
    CHECK(hipEventSynchronize(stop_share));
    
    float elapsed_time, elapsed_time_share;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    CHECK(hipEventElapsedTime(&elapsed_time_share, stop, stop_share));
    printf("Time_global = %g ms.\n", elapsed_time);
    printf("Time_share = %g ms.\n", elapsed_time_share);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));    

    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

    int ok = 1;
    for (int i = 0; i < m; ++i)
    { 
        for (int j = 0; j < k; ++j)
        {
            if(fabs(h_cs[i*k + j] - h_cc[i*k + j])>(1.0e-10))
            {
                printf("hcs: %d hc: %d  ",h_cs[i*k + j], h_c[i*k + j]);
                ok = 0;
            }
        }
    }
    
     printf("%d \n", h_cs[0]);
    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
    
    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    CHECK(hipHostFree(h_a));
    CHECK(hipHostFree(h_b));
    CHECK(hipHostFree(h_c));
    return 0;
}