#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[])
{
    int m=100;
    int n=2000;
    int k=100;

    int *h_a, *h_b, *h_c, *h_cc;
    CHECK(hipHostMalloc((void **) &h_a, sizeof(int)*m*n));
    CHECK(hipHostMalloc((void **) &h_b, sizeof(int)*n*k));
    CHECK(hipHostMalloc((void **) &h_c, sizeof(int)*m*k));
    CHECK(hipHostMalloc((void **) &h_cc, sizeof(int)*m*k));

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    int *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **) &d_a, sizeof(int)*m*n));
    CHECK(hipMalloc((void **) &d_b, sizeof(int)*n*k));
    CHECK(hipMalloc((void **) &d_c, sizeof(int)*m*k));

    // copy matrix A and B from host to device memory
    CHECK(hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice));

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);    

    hipMemcpy(h_c, d_c, (sizeof(int)*m*k), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    
    CHECK(hipMemcpy(h_c, d_c, (sizeof(int)*m*k), hipMemcpyDeviceToHost));

    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

    int ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if(fabs(h_cc[i*k + j] - h_c[i*k + j])>(1.0e-10))
            {
                
                ok = 0;
            }
        }
    }

    if(ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    CHECK(hipHostFree(h_a));
    CHECK(hipHostFree(h_b));
    CHECK(hipHostFree(h_c));
    return 0;
}


