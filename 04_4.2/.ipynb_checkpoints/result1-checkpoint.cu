#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 16
__managed__ int a[1000 * 1000];
__managed__ int b[1000 * 1000];
__managed__ int c_gpu[1000 * 1000];
__managed__ int c_cpu[1000 * 1000];

__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}
__global__ void gpu_matrix_mult_shared(int* d_a, int* d_b, int* d_result, int M, int N, int K)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub <= N/BLOCK_SIZE; ++sub)
    {
        int r = row;
        int c = sub * BLOCK_SIZE + threadIdx.x;
        idx = r * N + c;

        if (r >= M || c >= N)
        {
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        r = sub * BLOCK_SIZE + threadIdx.y;
        c = col;
        idx = r * K + c;
        if (c >= K || r >= N)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < M && col < K)
    {
        d_result[row * K + col] = tmp;
    }
}
void cpu_matrix_mult(int* a, int* b, int* h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += a[i * n + h] * b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const* argv[])
{
    int m = 1000;
    int n = 1000;
    int k = 1000;

    hipEvent_t start, stop_cpu, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = 0*rand() % 1024+1;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = 0 * rand() % 1024 +1;
        }
    }

    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);


    gpu_matrix_mult_shared << <dimGrid, dimBlock >> > (a, b, c_gpu, m, n, k);

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    cpu_matrix_mult(a, b, c_cpu, m, n, k);
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    float elapsed_time_cpu, elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    CHECK(hipEventElapsedTime(&elapsed_time_cpu, stop_gpu, stop_cpu));
    printf("GPU Time = %g ms.\n", elapsed_time_gpu);
    printf("CPU Time = %g ms.\n", elapsed_time_cpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_cpu));
    CHECK(hipEventDestroy(stop_gpu));

    

    int ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            //printf("GPU: % d; CPU: %d; ", h_c[i * k + j], h_cc[i * k + j]);
            if (fabs(c_gpu[i * k + j] - c_cpu[i * k + j]) > (1.0e-10))
            {

                ok = 0;
            }
            //printf("\n");
        }
    }

    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    return 0;
}