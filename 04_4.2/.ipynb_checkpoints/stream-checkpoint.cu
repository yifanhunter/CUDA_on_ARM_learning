#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)


__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}


int main( void ) {
    hipDeviceProp_t  prop;
    int whichDevice;
    CHECK( hipGetDevice( &whichDevice ) );
    CHECK( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    // start the timers
    CHECK( hipEventCreate( &start ) );
    CHECK( hipEventCreate( &stop ) );

    // initialize the streams
    CHECK( hipStreamCreate( &stream0 ) );
    CHECK( hipStreamCreate( &stream1 ) );

    // allocate the memory on the GPU
    CHECK( hipMalloc( (void**)&dev_a0, N * sizeof(int) ) );
    CHECK( hipMalloc( (void**)&dev_b0, N * sizeof(int) ) );
    CHECK( hipMalloc( (void**)&dev_c0, N * sizeof(int) ) );
    CHECK( hipMalloc( (void**)&dev_a1, N * sizeof(int) ) );
    CHECK( hipMalloc( (void**)&dev_b1, N * sizeof(int) ) );
    CHECK( hipMalloc( (void**)&dev_c1, N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    CHECK( hipHostAlloc( (void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );
    CHECK( hipHostAlloc( (void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );
    CHECK( hipHostAlloc( (void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    CHECK( hipEventRecord( start, 0 ) );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {
        // enqueue copies of a in stream0 and stream1
        CHECK( hipMemcpyAsync( dev_a0, host_a+i, N * sizeof(int), hipMemcpyHostToDevice, stream0 ) );
        CHECK( hipMemcpyAsync( dev_a1, host_a+i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1 ) );
        // enqueue copies of b in stream0 and stream1
        CHECK( hipMemcpyAsync( dev_b0, host_b+i, N * sizeof(int), hipMemcpyHostToDevice, stream0 ) );
        CHECK( hipMemcpyAsync( dev_b1, host_b+i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1 ) );

        kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
        kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 );

        CHECK( hipMemcpyAsync( host_c+i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0 ) );
        CHECK( hipMemcpyAsync( host_c+i+N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1 ) );
    }
    CHECK( hipStreamSynchronize( stream0 ) );
    CHECK( hipStreamSynchronize( stream1 ) );

    CHECK( hipEventRecord( stop, 0 ) );

    CHECK( hipEventSynchronize( stop ) );
    CHECK( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    CHECK( hipHostFree( host_a ) );
    CHECK( hipHostFree( host_b ) );
    CHECK( hipHostFree( host_c ) );
    CHECK( hipFree( dev_a0 ) );
    CHECK( hipFree( dev_b0 ) );
    CHECK( hipFree( dev_c0 ) );
    CHECK( hipFree( dev_a1 ) );
    CHECK( hipFree( dev_b1 ) );
    CHECK( hipFree( dev_c1 ) );
    CHECK( hipStreamDestroy( stream0 ) );
    CHECK( hipStreamDestroy( stream1 ) );

    return 0;
}

