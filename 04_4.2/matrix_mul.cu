#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 32
__managed__ int a[1000 * 1000];
__managed__ int b[1000 * 1000];
__managed__ int u_gpu[1000 * 1000];
__managed__ int u_cpu[1000 * 1000];

__global__ void gpu_matrix_mult_shared(int* d_a, int* d_b, int* d_result, int M, int N, int K)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub <= N/BLOCK_SIZE; ++sub)
    {
        int r = row;
        int c = sub * BLOCK_SIZE + threadIdx.x;
        idx = r * N + c;

        if (r >= M || c >= N)
        {
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        r = sub * BLOCK_SIZE + threadIdx.y;
        c = col;
        idx = r * K + c;
        if (c >= K || r >= N)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < M && col < K)
    {
        d_result[row * K + col] = tmp;
    }
}

void cpu_matrix_mult(int* a, int* b, int* h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h)
            {
                tmp += a[i * n + h] * b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const* argv[])
{
    int m = 1000;
    int n = 1000;
    int k = 1000;

	 //声明Event
    hipEvent_t start, stop_cpu, stop_gpu;
	
	 //创建Event
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventCreate(&stop_gpu));
	
	//开辟主机空间
//	  int *h_a, *h_b, *h_c, *h_cc;
//    CHECK(hipHostMalloc((void **) &h_a, sizeof(int)*m*n));
//    CHECK(hipHostMalloc((void **) &h_b, sizeof(int)*n*k));
//    CHECK(hipHostMalloc((void **) &h_c, sizeof(int)*m*k));
//    CHECK(hipHostMalloc((void **) &h_cc, sizeof(int)*m*k));

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = 0*rand() % 1024+1;
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = 0 * rand() % 1024 +1;
        }
    }
	
	
	//int *d_a, *d_b, *d_c;
    //CHECK(hipMalloc((void **) &d_a, sizeof(int)*m*n));
    //CHECK(hipMalloc((void **) &d_b, sizeof(int)*n*k));
    //CHECK(hipMalloc((void **) &d_c, sizeof(int)*m*k));

    // copy matrix A and B from host to device memory
    //CHECK(hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice));
    //CHECK(hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice));

	
	//开始start Event
    CHECK(hipEventRecord(start));
    //非阻塞模式
    hipEventQuery(start);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult_shared << <dimGrid, dimBlock >> > (a, b, u_gpu, m, n, k);
	//gpu_matrix_mult_shared<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k); 
	
	//CHECK(hipMemcpy(h_c, d_c, (sizeof(int)*m*k), hipMemcpyDeviceToHost));
    //hipDeviceSynchronize();

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

	// CPU 计算
    cpu_matrix_mult(a, b, u_cpu, m, n, k);
	//cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
	
	//记录时间消耗
    float elapsed_time_cpu, elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    CHECK(hipEventElapsedTime(&elapsed_time_cpu, stop_gpu, stop_cpu));
    printf("GPU Time = %g ms.\n", elapsed_time_gpu);
    printf("CPU Time = %g ms.\n", elapsed_time_cpu);

	//销毁Event
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_cpu));
    CHECK(hipEventDestroy(stop_gpu));

	//对比正确性
    int ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            if (fabs(u_gpu[i * k + j] - u_cpu[i * k + j]) > (1.0e-10))
            {
                ok = 0;
            }
            //printf("\n");
        }
    }

    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }
	
	// free memory
    //hipFree(d_a);
    //hipFree(d_b);
    //hipFree(d_c);
    //hipHostFree(h_a);
    //hipHostFree(h_b);
    //hipHostFree(h_c);
    //hipHostFree(h_cc);
    return 0;
}